#include "hip/hip_runtime.h"
// matrix bloom filter comparison
// author: candice quates
//
////////////////////////////////////////////////////////////////////////////
// Portions of this program were derived from cuda toolkit samples,
// specifically the shared-memory version of matrix multiplication,
// and the repeated range sample code.
//
// Those parts are subject to the nvidia EULA as below.
// 


///////////////////////////////////////////////////////////////////////////
// Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
///


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdint.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define MIN_FULLNESS 16

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples

// thrust for reductions -- smaller reductions are actually okay.
#include <hip/hip_runtime.h>

#include "matrixCompare.h"
#include <sdbf_class.h>
#include <bloom_filter.h>

#include <sdbf_set.h>
/**
 * Comparison (CUDA Kernel) on the device: C = popcll( A & B )
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void
CompKernelShared(uint16_t *C, uint64_t *A, uint64_t *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    uint64_t Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ uint64_t As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ uint64_t Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compare the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
			// checking for 0 less efficient than computation. less divergence.
			Csub += __popcll(As[ty][k] & Bs[k][tx]);
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

// kernel for minimum estimate cache
__global__ void EstCacheKernel(uint16_t *R) {
	double m = 2048;
	double k = 5;
	double exp = 1-1.0/2048;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	// bloom filter minimum estimate calculation for all element counts up to 256
	R[x*256+y]=(uint16_t)llrintf(m*(1- powf(exp,k*x) - powf(exp,k*y)+ powf(exp,k*(x+y))) );
	
}

// compute and apply cutoff based upon hamming weight and fullness of filters
// 1 and 2 apply to A and B-matrixes respectively.
__global__ void CutoffKernel(uint16_t *R, uint16_t *cache, uint16_t *s1, uint16_t *s2, uint16_t *ham1, uint16_t *ham2, int longsideB) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
	// if either filter does not have enough elements
	// throw out the result
	int min_est=0;
	int s1row=s1[row];
	int s2col=s2[col];
	if ((s1row < MIN_FULLNESS ) || (s2col < MIN_FULLNESS)) {
		R[row * longsideB + col]=0;
	} else {
		if (s1row==160 && s2col==160) // cache hit avoidance; most common case
			min_est=214;
		else 
			min_est=cache[s1row*256+s2col];	// expensive
		int max_est = (ham1[row] < ham2[col]) ? ham1[row]: ham2[col];
		float cut_off=(0.3*(float)(max_est-min_est)+(float)min_est);
		R[row * longsideB + col] = (R[row * longsideB + col] > cut_off)? (uint16_t)llrintf(100*(R[row * longsideB + col]-cut_off)/(max_est-cut_off)) : 0 ;
	}
}

// reduction kernel -- basic but still faster than reduce-by-key
// strided accesses etc probably would help.
__global__ void
ReduceKernel(uint16_t *A, uint16_t *C, int wA)
{
    // Each thread computes one element of C
	// from each row of A
    uint16_t Cmax = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < wA; ++e) {
		if (A[row * wA +e] > Cmax)
			Cmax = A[row * wA + e];
	}
    C[row] = Cmax;
}

void constantInitC(uint64_t *data, int size, uint64_t val)
{
    for (int i = 0; i < size; ++i)
        data[i] = val;
}

void constantInitCs(uint16_t *data, int size, uint16_t val)
{
    for (int i = 0; i < size; ++i)
        data[i] = val;
}
int nextpower2(int32_t bf_count) {
	int32_t v = bf_count;
	v--;
	v |= v >> 1;
	v |= v >> 2;
	v |= v >> 4;
	v |= v >> 8;
	v++;
	if (v < 32 ) v = 32;
	return v;
}

// wrapper bit for matrixCompare to compare sdbf sets
// 
int sdbfsetCompare(sdbf_set *refset, sdbf_set *target, bool quiet, int confidence) {    
    
	int shortside=32;

	// allocate these based on target sizing
	int longsidea=32768;
	int longsideb=32;

	int block_size=32;
	longsidea = nextpower2(refset->bf_vector->size());
	//longsidea *=4;
	if (!quiet)
	   std::cout <<"side A " << longsidea << std::endl;

    // Allocate host memory for matrices A and B
    unsigned int size_A = shortside *longsidea;
    unsigned int mem_size_A = sizeof(uint64_t) * size_A;
    uint64_t *h_A = (uint64_t *)malloc(mem_size_A);

	// Initialize host memory - ie make some data
    constantInitC(h_A, size_A, 0);
	uint8_t *h_A8;
	h_A8=(uint8_t*)h_A;
	uint16_t *ham_A = (uint16_t*)malloc(longsidea*sizeof(uint16_t));
    constantInitCs(ham_A, longsidea, 0);
	uint16_t *elem_A = (uint16_t*)malloc(longsidea*sizeof(uint16_t));
    constantInitCs(elem_A, longsidea, 0);
    // load bloom filters into A 
	for (int i=0; i < refset->bf_vector->size(); i++) {
		for (int j=0; j < 256; j++) {
			h_A8[256*i+j]=refset->bf_vector->at(i)->bf[j];
		}
		ham_A[i]=refset->bf_vector->at(i)->hamming;
		elem_A[i]=(uint16_t)refset->bf_vector->at(i)->elem_count();
	}
	


	unsigned int mem_size_R = longsidea * sizeof(uint16_t);
	uint16_t *results = (uint16_t *) malloc(mem_size_R);
	// B we will load repeatedly with one sdbf at a time
	// from target
    // allocate device memory for A parts
	// and transfer it up to the device.
	uint64_t *d_A, *d_B;
    hipError_t error;
    error = hipMalloc((void **) &d_A, mem_size_A);
    if (error != hipSuccess)    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    if (error != hipSuccess)    {
        printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	int mem_size_Act = longsidea*sizeof(uint16_t);
	uint16_t *ham_Ad, *elem_Ad;
	error = hipMalloc((void **) &elem_Ad, mem_size_Act);
	if (error != hipSuccess)    {
		printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}  	
	error = hipMalloc((void **) &ham_Ad, mem_size_Act);
	if (error != hipSuccess)    {
		printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}  	
	error = hipMemcpy(ham_Ad, ham_A, mem_size_Act, hipMemcpyHostToDevice);
	if (error != hipSuccess)    {
		printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	error = hipMemcpy(elem_Ad, elem_A, mem_size_Act, hipMemcpyHostToDevice);
	if (error != hipSuccess)    {
		printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	// for each sdbf in set2, load into B and compare it with A.
	for (int m=0; m < target->size(); m++) {
		longsideb=1024; // fix this
		longsideb = nextpower2(target->at(m)->filter_count());
		if (!quiet)
			std::cout <<"side B " << longsideb << std::endl;
		unsigned int size_B = shortside *longsideb;
		unsigned int mem_size_B = sizeof(uint64_t) * size_B;
		uint64_t *h_B = (uint64_t *)malloc(mem_size_B);
		constantInitC(h_B, size_B, 0);
		uint8_t *h_B8;
		h_B8=(uint8_t*)h_B;    
		
		int mem_size_Bct =sizeof(uint16_t)*longsideb;
		
		uint16_t *ham_B = (uint16_t*)malloc(mem_size_Bct);
		constantInitCs(ham_B, longsideb, 0);
		uint16_t *elem_B = (uint16_t*)malloc(mem_size_Bct);
		constantInitCs(elem_B, longsideb, 0);
		uint8_t *tmpbuf;
		int max_elem=0;
	    for (int j=0; j < target->at(m)->filter_count() ; j++) {
			ham_B[j]=target->at(m)->hamming[j];
			elem_B[j]=sdbf::get_elem_count(target->at(m),j);
			if (elem_B[j] > max_elem) max_elem=elem_B[j];			                  
            for (int i=0; i< 256; i+=8) {
                for (int k=0;k<8;k++)
					h_B8[i*longsideb+k+j*8]=target->at(m)->buffer[i+k+j*256];
            }
		}
		// if the whole block has too few elements, skip it.
		// for filtering out empty parts of drives
		if (max_elem < MIN_FULLNESS) {
			free(h_B);
			free(ham_B);
			free(elem_B);
			continue;
		}

		uint16_t *ham_Bd, *elem_Bd;
		error = hipMalloc((void **) &elem_Bd, mem_size_Bct);
		if (error != hipSuccess)    {
			printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}  	
		error = hipMalloc((void **) &ham_Bd, mem_size_Bct);
		if (error != hipSuccess)    {
			printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}  	
		error = hipMemcpy(ham_Bd, ham_B, mem_size_Bct, hipMemcpyHostToDevice);
		if (error != hipSuccess)    {
			printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}
		error = hipMemcpy(elem_Bd, elem_B, mem_size_Bct, hipMemcpyHostToDevice);
		if (error != hipSuccess)    {
			printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}

		error = hipMalloc((void **) &d_B, mem_size_B);
		if (error != hipSuccess)    {
			printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}  	
		error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
		if (error != hipSuccess)    {
			printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
			exit(EXIT_FAILURE);
		}
		int matrix_result;
		matrix_result = matrixCompare(block_size, shortside, longsidea,longsideb, d_A, d_B, results, elem_Ad, elem_Bd, ham_Ad, ham_Bd);
		int result_count=0;
		int score=0;
		int filter_count=0;
		if (!quiet)
		    std::cout << target->at(m)->name()  << std::endl;
		int setsize =refset->bf_vector->size();
		for (int i=0; i < setsize; i++) {
			if (results[i]>0) { 
			    result_count++;
			    score+=results[i];
			}
			if (elem_A[i]>=MIN_FULLNESS)
				filter_count++;
			if (i+1 == setsize) {
				if (result_count!=0  && (score/filter_count >= confidence)) {
					cout << target->at(m)->name()  << "|"<< refset->bf_vector->at(i)->name() << "|" << score/filter_count<< endl;
					result_count=0;
					score=0;
				}
			} else if (refset->bf_vector->at(i)->bloom_id() != refset->bf_vector->at(i+1)->bloom_id()) {
				if (result_count!=0 && (score/filter_count >= confidence)) {	
					cout << target->at(m)->name()  << "|"<< refset->bf_vector->at(i)->name() << "|" << score/filter_count<< endl;		
				}
				filter_count=0;
				result_count=0;
				score=0;
			}
		}	

		free(h_B);
		free(ham_B);
		free(elem_B);
		hipFree(d_B);
		hipFree(ham_Bd);
		hipFree(elem_Bd);
	}
	hipFree(d_A);
	free(ham_A);
	free(elem_A);
	free(h_A);
	hipFree(ham_Ad);
	hipFree(elem_Ad);
	free(results);
	return 0;
}

// Compares two matrices in device memory.
// host required to pre-allocate result memory in *resptr
int matrixCompare(int block_size, int shortside, int long_A, int long_B, uint64_t *d_A, uint64_t *d_B, uint16_t *resptr, 
	uint16_t *elem_A, uint16_t *elem_B,uint16_t *ham_A,uint16_t *ham_B)
{
	
    dim3 dimsA(shortside,long_A, 1);
    dim3 dimsB(long_B, shortside, 1);
    // size calcs
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(uint64_t) * size_A;
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(uint64_t) * size_B;

    // Size  matrix C 
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int size_C = dimsC.x * dimsC.y;
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(uint16_t);
	
    unsigned int mem_size_R = dimsA.y * sizeof(uint16_t);
    //uint16_t *h_C = (uint16_t *) malloc(mem_size_C);
	
    // Declare device memory
    uint16_t *d_C;
	
    // allocate device memory for computation matrix
    hipError_t error;
    error = hipMalloc((void **) &d_C, mem_size_C);
    if (error != hipSuccess)    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	uint16_t *d_cache;
	error = hipMalloc((void **) &d_cache, 256*256*(sizeof(uint16_t)));
	if (error != hipSuccess)    {
		printf("hipMalloc est_cache returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// compute cutoff cache values
	dim3 threadscache(16,16);
	dim3 gridcache(256/threadscache.x,256/threadscache.y);
	EstCacheKernel<<<gridcache,threadscache>>>(d_cache);   
    // Setup thread block and grid
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);
    // Compare Kernel 
    CompKernelShared<32><<< grid, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	// cutoff kernel
	CutoffKernel<<<grid,threads>>>(d_C, d_cache, elem_A, elem_B, ham_A, ham_B, dimsC.x);

	// allocate result space
	uint16_t *d_R;
	error = hipMalloc((void **) &d_R, mem_size_R);
    if (error != hipSuccess)    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	//reduction kernel -- runs on 1-dimension not in a grid shape like the others
	//ReduceKernel<<<dimsC.y/1024,1024>>>(d_C, d_R, dimsC.x);
	ReduceKernel<<<dimsC.y/16,16>>>(d_C, d_R, dimsC.x);
    error = hipMemcpy(resptr, d_R, mem_size_R, hipMemcpyDeviceToHost);
    if (error != hipSuccess)    {
        printf("hipMemcpy (h_R,d_R) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }    	

	//for (int i=0; i < dimsC.y; i++) {
	//	cout << resptr[i] << " " ;
	//}	
    
    hipFree(d_C);
	hipFree(d_R);
	hipFree(d_cache);
    return EXIT_SUCCESS;
    
}

int deviceSetup(int devID) {
    hipSetDevice(devID);
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)    {
        fprintf(stderr,"hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.computeMode == hipComputeModeProhibited)    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        return(1);
    }

    if (error != hipSuccess)    {
        fprintf(stderr,"hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else    {
        fprintf(stderr,"GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
	return 0;
}

int deviceTeardown() {
	hipDeviceReset(); // at end?
	return 0;
}
